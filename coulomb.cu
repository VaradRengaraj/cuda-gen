#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <stdlib.h>
#include <iostream>

__global__ void coulombMatrix(double *pos, double *col, int *chargeptr, int nx, int ny, int cutoff, double bc)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy*nx + ix;
    double rd;

    if(ix < nx && iy < ny)
    {
        if(ix == iy){
            *(col+idx) = 0.5*pow(chargeptr[ix], 2.4);
        }
        else if(ix > iy){
            rd = pow(pow(*(pos+(iy*3))-*(pos+(ix*3))-bc*round((*(pos+(iy*3))-*(pos+(ix*3)))/bc),2)+
                pow(*(pos+(iy*3)+1)-*(pos+(ix*3)+1)-bc*round((*(pos+(iy*3)+1)-*(pos+(ix*3)+1))/bc),2)+
                pow(*(pos+(iy*3)+2)-*(pos+(ix*3)+2)-bc*round((*(pos+(iy*3)+2)-*(pos+(ix*3)+2))/bc),2),0.5);
            //printf("rd %lf\n", rd);
            if(rd >= cutoff){
                *(col+idx) = 0;
            }
            else{
                *(col+idx) = (chargeptr[ix]*chargeptr[iy])/rd;
            }
        }
        else{}
    }
}

__global__ void coulombMatrixLT(double *col, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy*nx + ix;
    unsigned int idr = ix*nx + iy;

    if(ix < nx && iy < ny)
    {
        if(ix < iy)
        {
            *(col+idx) = *(col+idr);
        }
    }
}

