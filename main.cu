// System includes

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<pthread.h>
#include<unistd.h>
#include<limits.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>

// DEFINES 
#define NUM_ATOMS 81
#define DEBUG 0
#define ERROR(msg)\
    fprintf(stderr, "%s,%d ", __func__, __LINE__);\
    fprintf(stderr, "%s", msg);
#define PLACE_HOLDER1 4
#define PLACE_HOLDER2 30
#define PLACE_HOLDER3 56
#define LINE_SPLIT_SIZE 25


extern __global__ void coulombMatrix(double *pos, double *col, int *chargeptr, int nx, int ny, int cutoff, double bc);
extern __global__ void coulombMatrixLT(double *col, int nx, int ny);
extern __global__ void jacobi(double *arr_ptr, int *pair_arr, int n, int *cont, double tolerance);
extern __global__ void copysubmat(double *subm, int N, int num);
extern __global__ void submatrix(double *col, int nx, int *submatsizes);

pthread_mutex_t cond_var_lock = PTHREAD_MUTEX_INITIALIZER;
pthread_cond_t cond_var = PTHREAD_COND_INITIALIZER;
pthread_mutex_t crit_lock = PTHREAD_MUTEX_INITIALIZER;

char *frame_bufs[10] = {0,};
unsigned int frame_size;
unsigned int line1_size;
unsigned int line2_size;
unsigned int line3_size;

#define NTHREADS 3

//cudaStream_t streams[NTHREADS];

/*
 * Function that parses a position frame, creates the coulomb matrix, spawns submatrices and calculates eigen values
 * for each submatrix
 */
void *main_job_cuda(void *count)
{
    // parser for each frame
    double frame[NUM_ATOMS][3];
    unsigned int j = 0;
    char temp1[LINE_SPLIT_SIZE], temp2[LINE_SPLIT_SIZE], temp3[LINE_SPLIT_SIZE];
    unsigned int cnt = *(int *)count;
    char *buf;
    unsigned int i;
    int den_1, den_2, den_3;
    hipError_t status;
    double *posptr;
    double col[NUM_ATOMS][NUM_ATOMS] = {0,};
    double *colptr, *bufptr, *submat1;
    int *chargptr;
    int charge[3] = {8, 1, 1};
    int charge_arr[NUM_ATOMS] = {0,};
    //int i,j;
    int rcut = 5;
    int dimx = 12;
    int dimy = 12;
    int nx = NUM_ATOMS;
    int ny = NUM_ATOMS;
    int submatsize[NUM_ATOMS] = {0,};
    int *submat;
    dim3 block(dimx, dimy);
    dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);
    double bc = 9.3214112017424995;
    int n, cont=1;
    //int *pair = (int*)malloc(n*sizeof(int));
    double tolerance = 0.000000000001;
    int dim;
    int *d_cont;
 
    den_1 = 0;
    den_2 = 0;
    den_3 = 0;
    printf("\n main job thread # %d", cnt);
    //cudaStreamCreate(&streams[cnt]);

    // nth thread picks nth frame
    buf = frame_bufs[0] + cnt * frame_size + line1_size + line2_size;

    //loop which parses the frames x, y, z coordinates and stores in an array
    // terms such as 'E-003' are taken care 
    while(j < NUM_ATOMS){
        memcpy(temp1, buf + PLACE_HOLDER1, LINE_SPLIT_SIZE);
        memcpy(temp2, buf + PLACE_HOLDER2, LINE_SPLIT_SIZE);
        memcpy(temp3, buf + PLACE_HOLDER3, LINE_SPLIT_SIZE);

	if(temp1[20] == 'E')
	    den_1 = temp1[24] - '0';

        if(temp2[20] == 'E')
	    den_2 = temp2[24] - '0';

        if(temp3[20] == 'E')
	    den_3 = temp3[24] - '0';
	    
        sscanf(temp1, "%lf", &frame[j][0]);
        sscanf(temp2, "%lf", &frame[j][1]);
        sscanf(temp3, "%lf", &frame[j][2]);
 
        if(!den_1)
	    frame[j][0] /= pow(10, den_1);

        if(!den_2)
	    frame[j][1] /= pow(10, den_2);

        if(!den_3)
	    frame[j][2] /= pow(10, den_3);

        j += 1;
        buf += line3_size;
    }

    #if DEBUG
    if(cnt == 49){
    for(j = 0; j < NUM_ATOMS; j++){
        for(i = 0; i < 3; i++){
            printf(" %.17g", frame[j][i]);
            }
        printf("\n");
    }}
    #endif

    // coloumb matrix creation
    j = 0;
    for(i=0; i<(sizeof(charge_arr)/sizeof(int)); i++)
    {
        charge_arr[i] = charge[j];
        j++;
        if(j == 3)
            j=0;
    }

    for(i=0; i<(sizeof(charge_arr)/sizeof(int)); i++)
    {
        printf(" %d", charge_arr[i]);
    }

    // allocate memory for the frame in the gpu
    status = hipMalloc((double **)&posptr, NUM_ATOMS*3*sizeof(double));
    if( status != hipSuccess) {
        fprintf(stderr, " Could not allocate memory on the device!!");
    }

    // allocate memory for the coulomb matrix in the gpu
    status = hipMalloc((double **)&colptr, NUM_ATOMS*NUM_ATOMS*sizeof(double));
    if( status != hipSuccess) {
        fprintf(stderr, " Could not allocate memory on the device!!");
    }

    // allocate memory for the charge array in the gpu
    status = hipMalloc((int **)&chargptr, NUM_ATOMS*sizeof(int));
    if( status != hipSuccess) {
        fprintf(stderr, " Could not allocate memory on the device!!");
    }

    // allocate memory for submat in the gpu
    // submat contains necessary information to create submatrices for the coulomb matrix
    status = hipMalloc((int **)&submat, NUM_ATOMS*sizeof(int));
    if( status != hipSuccess) {
        fprintf(stderr, " Could not allocate memory on the device!!");
    }

    // frame in copied from main memory to gpu memory
    status = hipMemcpy(posptr, frame, NUM_ATOMS*3*sizeof(double), hipMemcpyHostToDevice);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the position array to the device!!");
    }

    // coulomb matrix in the gpu is zero initialized
    status = hipMemcpy(colptr, col, NUM_ATOMS*NUM_ATOMS*sizeof(double), hipMemcpyHostToDevice);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the position array to the device!!");
    }

    // charge array is copied from main memory to gpu memory
    status = hipMemcpy(chargptr, charge_arr, NUM_ATOMS*sizeof(int), hipMemcpyHostToDevice);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the position array to the device!!");
    }

    // !! invoke the coulomb matrix creation cuda kernel !!
    coulombMatrix <<< grid, block >>>(posptr, colptr, chargptr, nx, ny, rcut, bc);
    hipDeviceSynchronize();
    // !! invoke the coulomb matrix lower triangler populate kernel !!
    coulombMatrixLT <<< grid, block >>>(colptr, nx, ny);
    hipDeviceSynchronize();

    status = hipMemcpy(col, colptr, NUM_ATOMS*NUM_ATOMS*sizeof(double), hipMemcpyDeviceToHost);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the position array to the host!!");
    }

    #if DEBUG
    //if(cnt == 49){
    //print the composed coulomb matrix
    printf("\n couloumb matrix thread #%d\n", cnt);
    for(i=0; i<NUM_ATOMS; i++)
    {
        printf("\n");
        for(j=0; j<NUM_ATOMS; j++)
        {
            printf(" %lf",col[i][j]);
        }
        //printf("\n");
    }
    //}
    #endif

    // memory intensive operation starts here. allowing only 1 thread from here to create submatrices and eigen values.
    pthread_mutex_lock(&crit_lock);

    // copy the coulomb matrix to the gpu memory
    status = hipMemcpy(colptr, col, NUM_ATOMS*NUM_ATOMS*sizeof(double), hipMemcpyHostToDevice);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the coulomb matrix array to the host!!");
    }

    //submatrix creation
    submatrix <<< 1, 100 >>>(colptr, nx, submat);
    hipDeviceSynchronize();

    //copy submat matrix to the host from gpu
    // submat is an array of NUM_ATOMS size where every ith element contains the number of zeros in the ith row/col.
    status = hipMemcpy(submatsize, submat, NUM_ATOMS*sizeof(int), hipMemcpyDeviceToHost);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the submat sizes array to the host!!");
    }

    #if DEBUG
    printf("\n\n first submat half size %d\n\n", submatsize[0]);
    #endif 

    int num = 0;

    printf(" comes in #%d", cnt);

    while(num < NUM_ATOMS){
        //verify the working of jacobi eigen solver for the first submatrix
        status = hipMalloc((double **)&submat1, submatsize[num]*submatsize[num]*sizeof(double));
        if( status != hipSuccess) {
            fprintf(stderr, " Could not allocate memory on the device!!");
        }

        bufptr = (double *)malloc(submatsize[num]*submatsize[num]*sizeof(double));
        if(!bufptr)
            fprintf(stderr, " Could not allocate memory for submatrix!!");

        copysubmat <<< 1, 1 >>>(submat1, submatsize[num], num);

        status = hipMemcpy(bufptr, submat1, submatsize[num]*submatsize[num]*sizeof(double), hipMemcpyDeviceToHost);
        if( status != hipSuccess) {
            fprintf(stderr, " Could not copy the submat sizes array to the host!!");
        }

        #if DEBUG
        printf("\n submatrix \n");
        //print the 1st submatrix
        for(i=0; i< submatsize[0]; i++){
            for(j=0; j<submatsize[0]; j++){
                printf(" %lf", *(bufptr+i*submatsize[0]+j));
            }
            printf("\n");
        }
        #endif

        hipMalloc((void**) &d_cont, sizeof(int));
        hipMemcpy(d_cont, &cont, sizeof(int), hipMemcpyHostToDevice);

        dim = submatsize[num];

        if(dim % 2 == 0){
            n = dim;
        }
        else
            n = dim + 1;

        int *pair = (int*)malloc(n*sizeof(int));
        int *d_pair;
        status = hipMalloc( (void**) &d_pair, n*sizeof(int));
        if( status != hipSuccess) {
            fprintf(stderr, " Could not allocate pair memory on the device!!");
        }

        if(dim % 2 == 0){
            /*initializing pair matrix*/
            for (i = 0; i < n; i++)
                *(pair + i) = i;
        }
        else{
            for (i = 0; i < dim; i++)
                *(pair + i) = i;
            *(pair + n - 1) = 999;
        }
        printf("\n n %d\n", n);

        status = hipMemcpy(submat1, bufptr, submatsize[num]*submatsize[num]*sizeof(double), hipMemcpyHostToDevice);
        if( status != hipSuccess) {
            fprintf(stderr, " Could not copy the submat array to the host!!");
        }

        status = hipMemcpy(d_pair, pair, n*sizeof(int), hipMemcpyHostToDevice);
        if( status != hipSuccess) {
            fprintf(stderr, " Could not copy the pair array to the host!!");
        }

        jacobi<<<1, n/2>>>(submat1, d_pair, submatsize[num], d_cont, tolerance);

        hipMemcpy(bufptr, submat1, submatsize[num]*submatsize[num]*sizeof(double), hipMemcpyDeviceToHost);

        printf("\n\n eigen values here for #%d", cnt);
        //print the 1st submatrix
        for(i=0; i< submatsize[num]; i++){
            for(j=0; j<submatsize[num]; j++){
	      if(i == j)
                  printf(" %lf", *(bufptr+i*submatsize[num]+j));
              }
            printf("\n");
        }
        num += 1;
        hipFree(d_pair);
        hipFree(d_cont);
        hipFree(submat1);
        free(bufptr);
        free(pair);
    }

    pthread_mutex_unlock(&crit_lock);
    hipFree(posptr);
    hipFree(colptr);
    hipFree(chargptr);
}

/**
 * Parse the data
 */

void *parse_pos_file(void *arg)
{
    unsigned int i = 0;
    int count = 0;
    pthread_t threads[NTHREADS];
    void * retvals[NTHREADS];
    int *cnt;

    printf(" Thread 2");
    // cond_wait
    pthread_mutex_lock(&cond_var_lock);

    while(frame_bufs[0] == 0)
        pthread_cond_wait(&cond_var, &cond_var_lock);

    pthread_mutex_unlock(&cond_var_lock);

    //printf("\n comes here --1");
    //printf("\n");
    //buf = frame_bufs[0];
    //for(i = 0; i < 100; i++)
    //    printf("%c", *(buf+i));

   
    // launch 50 threads which parses the pos frame buffer and performs cuda operations in parallel.
    // each of the thread creates the coulomb matrix, does submatrix reductions and computes eigen values.
    // the eigen values are finally written as hdf5 files.
    for(count = 0; count < NTHREADS; count++)
    {
        fflush(stdout);
        cnt = (int *)malloc(1*sizeof(int));
        *cnt = count;
        if(pthread_create(&threads[count], NULL, main_job_cuda, (void *)cnt) != 0)
	{
	    printf("error: cannot create thread # %d\n", *cnt);
	    return (void *)NULL;
	}
    }

    for(i = 0; i < NTHREADS; i++)
    {
        if(pthread_join(threads[i], &retvals[i]) != 0)
	{
	    printf("error: cannot join thread # %d\n", i);
	    return (void *)NULL;
	}
    }

}

/*
 * Reads the position file
 */

void *read_pos_file(void *pth)
{
    char *path = (char *)pth;
    FILE *fp;
    char buf[256] = {0,};
    unsigned int i = 0;
    //unsigned int frame_size = 0;
    char *buff = NULL;
    printf("\n File path is %s", path);
    int nItemsread;

    fp = fopen(path, "r");

    if(fp == NULL){
        ERROR(" File open error!! \n");
        return NULL;
    }
    // estimate the memory size needed for a frame from pos file. Reads the first three lines.
    while(fgets(buf, 256, (FILE *)fp) != NULL){
        //printf("strlen(buf) is %d", strlen(buf));
	i++;

	if(i == 1){
	    frame_size += strlen(buf);
	    line1_size = strlen(buf);
	}
	else if(i == 2){
	frame_size += strlen(buf);
	line2_size = strlen(buf);
	}
	else{
	    frame_size += NUM_ATOMS * strlen(buf);
	    line3_size = strlen(buf);
        }
	if(i == 3)
	    break;
    }

#if DEBUG
    for(i = 0; i < 10; i++)
        printf("%c", buf[i]);
#endif

    printf("memory size req for a frame is %d", frame_size); 
    fseek(fp, 0, SEEK_SET);
    // consumer thread (thread 2) should wait till the data is ready
    // file data is read in chunks and each chunk is of the size of 50 frames
    // this helps in deallocating the memory once the processing is done
    // first 50 frames from the pos file is read and condition is signalled to thread 2
    pthread_mutex_lock(&cond_var_lock);
    buff = (char *)malloc(frame_size * 50 * sizeof(char));

    if(buff == NULL){
        ERROR("Memory allocation failed!! \n");
        return NULL;
    }

    nItemsread = fread(buff, sizeof(char), frame_size * 50, fp);
 
    if(nItemsread != frame_size * 50){
        ERROR("Req number of frame data is not in the pos file! \n");
        return NULL;
    }

    frame_bufs[0] = buff;
    pthread_cond_signal(&cond_var);
    pthread_mutex_unlock(&cond_var_lock);

    for(i = 1; i < 10; i++)
    {
        buff = (char *)malloc(frame_size * 50 * sizeof(char));

        if(buff == NULL){
	    ERROR("Memory allocation failed!! \n");
            return NULL;
        }

        nItemsread = fread(buff, sizeof(char), frame_size * 50, fp);

        if(nItemsread != frame_size * 50){
	    ERROR("Req number of frame data is not in the pos file! \n");
            return NULL;
        }

        frame_bufs[i] = buff;
    }

    fclose(fp);
}

/**
 * Program main
 */

int main(int argc, char *argv[])
{
    pthread_t thread1, thread2;
    int ret1, ret2; 
    char pos_file_name[] = "pos.xyz";
    char cwd[PATH_MAX];
    char file_path[PATH_MAX + strlen(pos_file_name)];

    // File path, where the pos, frc and ener file is located should be given
    if(argc == 1)
    {    
        printf("Program expects the directory name where pos/frc/ener file is located");
        return -1;
    }

    if (getcwd(cwd, sizeof(cwd)) != NULL) {
        printf("Current working dir: %s\n", cwd);
    } 
    else {
        perror("getcwd() error");
        return -1;
    }

    //printf("cwd is %s", cwd);
    strcat(file_path, cwd);
    strcat(file_path, "/");
    strcat(file_path, argv[1]);
    strcat(file_path, "/");
    strcat(file_path, pos_file_name);
    printf("\n file_path is %s", file_path);

    // thread1 reads the pos file
    // thread2 parses the pos file
    ret1 = pthread_create( &thread1, NULL, read_pos_file, (void *)file_path);     
    ret2 = pthread_create( &thread2, NULL, parse_pos_file, (void *)NULL);

    pthread_join(thread1, NULL);
    pthread_join(thread2, NULL);

    return 0;
}
