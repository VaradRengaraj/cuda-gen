
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<pthread.h>
#include<unistd.h>
#include<limits.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>


#define NUM_ATOMS 81

extern __global__ void coulombMatrix(double *pos, double *col, int *chargeptr, int nx, int ny, int cutoff, double bc);
extern __global__ void coulombMatrixLT(double *col, int nx, int ny);
extern __global__ void jacobi(double *arr_ptr, int *pair_arr, int n, int *cont, double tolerance);
extern __global__ void copysubmat(double *subm, int N, int num);
extern __global__ void submatrix(double *col, int nx, int *submatsizes);

pthread_mutex_t cond_var_lock = PTHREAD_MUTEX_INITIALIZER;
pthread_cond_t cond_var = PTHREAD_COND_INITIALIZER;
pthread_mutex_t crit_lock = PTHREAD_MUTEX_INITIALIZER;

char *frame_bufs[10] = {0,};
unsigned int frame_size;
unsigned int line1_size;
unsigned int line2_size;
unsigned int line3_size;

#define NTHREADS 3

//cudaStream_t streams[NTHREADS];

void *main_job_cuda(void *count)
{

// parser for each frame
    double frame[NUM_ATOMS][3];
    unsigned int j = 0;
    char temp1[25], temp2[25], temp3[25];
    unsigned int cnt = *(int *)count;
    char *buf;
    unsigned int i;
    int den_1, den_2, den_3;
    hipError_t status;
    double *posptr;
    double col[NUM_ATOMS][NUM_ATOMS] = {0,};
    double *colptr, *bufptr, *submat1;
    int *chargptr;
    int charge[3] = {8, 1, 1};
    int charge_arr[NUM_ATOMS] = {0,};
    //int i,j;
    int rcut = 5;
    int dimx = 12;
    int dimy = 12;
    int nx = NUM_ATOMS;
    int ny = NUM_ATOMS;
    int submatsize[NUM_ATOMS] = {0,};
    int *submat;
    dim3 block(dimx, dimy);
    dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);
    double bc = 9.3214112017424995;
    int n, cont=1;
    //int *pair = (int*)malloc(n*sizeof(int));
    double tolerance = 0.000000000001;
    int dim;
    int *d_cont;
 
    //if(cnt == 0)
    //    return NULL;
  
    den_1 = 0;
    den_2 = 0;
    den_3 = 0;
    //fflush(stdout);
    //printf(" val %d ", *(int *)count);
    printf("\n main job thread # %d", cnt);
    //cudaStreamCreate(&streams[cnt]);
    buf = frame_bufs[0] + cnt * frame_size + line1_size + line2_size;
    //i += line1_size + line2_size;
    while(j < NUM_ATOMS){
        memcpy(temp1, buf + 4, 25);
        memcpy(temp2, buf + 30, 25);
        memcpy(temp3, buf + 56, 25);
        //for(i = 0; i < 18; i++)
        //    printf(" %c", temp3[i]);
        //printf("\n");
	if(temp1[20] == 'E')
	    den_1 = temp1[24] - '0';

        if(temp2[20] == 'E')
	    den_2 = temp2[24] - '0';

        if(temp3[20] == 'E')
	    den_3 = temp3[24] - '0';
	    
        sscanf(temp1, "%lf", &frame[j][0]);
        sscanf(temp2, "%lf", &frame[j][1]);
        sscanf(temp3, "%lf", &frame[j][2]);
 
        if(!den_1)
	    frame[j][0] /= pow(10, den_1);

        if(!den_2)
	    frame[j][1] /= pow(10, den_2);

        if(!den_3)
	    frame[j][2] /= pow(10, den_3);

        j += 1;
        buf += line3_size;
    }

#if 1
    if(cnt == 49){
    for(j = 0; j < NUM_ATOMS; j++){
        for(i = 0; i < 3; i++){
            printf(" %.17g", frame[j][i]);
            }
        printf("\n");
    }}
#endif

// coloumb matrix creation
    j = 0;
    for(i=0; i<(sizeof(charge_arr)/sizeof(int)); i++)
    {
        charge_arr[i] = charge[j];
        j++;
        if(j == 3)
            j=0;
    }
    for(i=0; i<(sizeof(charge_arr)/sizeof(int)); i++)
    {
        printf(" %d", charge_arr[i]);
    }

    status = hipMalloc((double **)&posptr, NUM_ATOMS*3*sizeof(double));
    //status = cudaMalloc((double **)&posptr, NUM_ATOMS*3*sizeof(double),stream[cnt]);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not allocate memory on the device!!");
    }

    status = hipMalloc((double **)&colptr, NUM_ATOMS*NUM_ATOMS*sizeof(double));
    //status = cudaMalloc((double **)&colptr, NUM_ATOMS*NUM_ATOMS*sizeof(double),stream[cnt]);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not allocate memory on the device!!");
    }

    status = hipMalloc((int **)&chargptr, NUM_ATOMS*sizeof(int));
    //status = cudaMalloc((int **)&chargptr, NUM_ATOMS*sizeof(int),stream[cnt]);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not allocate memory on the device!!");
    }

    status = hipMalloc((int **)&submat, NUM_ATOMS*sizeof(int));
    //status = cudaMalloc((int **)&submat, NUM_ATOMS*sizeof(int),stream[cnt]);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not allocate memory on the device!!");
    }

    status = hipMemcpy(posptr, frame, NUM_ATOMS*3*sizeof(double), hipMemcpyHostToDevice);
    //status = cudaMemcpy(posptr, frame, NUM_ATOMS*3*sizeof(double), cudaMemcpyHostToDevice,stream[cnt]);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the position array to the device!!");
    }

    status = hipMemcpy(colptr, col, NUM_ATOMS*NUM_ATOMS*sizeof(double), hipMemcpyHostToDevice);
    //status = cudaMemcpy(colptr, col, NUM_ATOMS*NUM_ATOMS*sizeof(double), cudaMemcpyHostToDevice,stream[cnt]);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the position array to the device!!");
    }

    status = hipMemcpy(chargptr, charge_arr, NUM_ATOMS*sizeof(int), hipMemcpyHostToDevice);
    //status = cudaMemcpy(chargptr, charge_arr, NUM_ATOMS*sizeof(int), cudaMemcpyHostToDevice,stream[cnt]);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the position array to the device!!");
    }

    coulombMatrix <<< grid, block >>>(posptr, colptr, chargptr, nx, ny, rcut, bc);
    //coulombMatrix <<< grid, block, 0, streams[cnt] >>>(posptr, colptr, chargptr, nx, ny, rcut, bc);
    hipDeviceSynchronize();
    //cudaStreamSynchronize(0);
    coulombMatrixLT <<< grid, block >>>(colptr, nx, ny);
    //coulombMatrixLT <<< grid, block, 0, streams[cnt] >>>(colptr, nx, ny);
    hipDeviceSynchronize();
    //cudaStreamSynchronize(0);

    status = hipMemcpy(col, colptr, NUM_ATOMS*NUM_ATOMS*sizeof(double), hipMemcpyDeviceToHost);
    //status = cudaMemcpy(col, colptr, NUM_ATOMS*NUM_ATOMS*sizeof(double), cudaMemcpyDeviceToHost, stream[cnt]);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the position array to the host!!");
    }

    //if(cnt == 49){
    //print the composed coulomb matrix
    printf("\n couloumb matrix thread #%d\n", cnt);
    for(i=0; i<NUM_ATOMS; i++)
    {
        printf("\n");
        for(j=0; j<NUM_ATOMS; j++)
        {
            printf(" %lf",col[i][j]);
        }
        //printf("\n");
    }
    //}

#if 1
    pthread_mutex_lock(&crit_lock);

    status = hipMemcpy(colptr, col, NUM_ATOMS*NUM_ATOMS*sizeof(double), hipMemcpyHostToDevice);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the coulomb matrix array to the host!!");
    }

//submatrix creation
    submatrix <<< 1, 100 >>>(colptr, nx, submat);
    //submatrix <<< 1, 100, 0, streams[cnt] >>>(colptr, nx, submat);
    hipDeviceSynchronize();
    //cudaStreamSynchronize(0);

    status = hipMemcpy(submatsize, submat, NUM_ATOMS*sizeof(int), hipMemcpyDeviceToHost);
    if( status != hipSuccess) {
        fprintf(stderr, " Could not copy the submat sizes array to the host!!");
    }

    printf("\n\n first submat half size %d\n\n", submatsize[0]);

    int num = 0;

    //pthread_mutex_lock(&crit_lock);
    printf(" comes in #%d", cnt);

    while(num < NUM_ATOMS){
        //verify the working of jacobi eigen solver for the first submatrix
        status = hipMalloc((double **)&submat1, submatsize[num]*submatsize[num]*sizeof(double));
        if( status != hipSuccess) {
            fprintf(stderr, " Could not allocate memory on the device!!");
        }

        bufptr = (double *)malloc(submatsize[num]*submatsize[num]*sizeof(double));
        if(!bufptr)
            fprintf(stderr, " Could not allocate memory for submatrix!!");

        copysubmat <<< 1, 1 >>>(submat1, submatsize[num], num);
	//copysubmat <<< 1, 1, 0, streams[cnt] >>>(submat1, submatsize[num], num);
        //cudaDeviceSynchronize();
	//cudaStreamSynchronize(0);

        status = hipMemcpy(bufptr, submat1, submatsize[num]*submatsize[num]*sizeof(double), hipMemcpyDeviceToHost);
        if( status != hipSuccess) {
            fprintf(stderr, " Could not copy the submat sizes array to the host!!");
        }

        //printf("\n 1st submatrix \n");
        //print the 1st submatrix
        //for(i=0; i< submatsize[0]; i++){
        //    for(j=0; j<submatsize[0]; j++){
        //        printf(" %lf", *(bufptr+i*submatsize[0]+j));
        //        }
        //    printf("\n");
        //}

        hipMalloc((void**) &d_cont, sizeof(int));
        hipMemcpy(d_cont, &cont, sizeof(int), hipMemcpyHostToDevice);

        dim = submatsize[num];

        if(dim % 2 == 0){
            n = dim;
        }
        else
            n = dim + 1;

        int *pair = (int*)malloc(n*sizeof(int));
        int *d_pair;
        status = hipMalloc( (void**) &d_pair, n*sizeof(int));
        if( status != hipSuccess) {
            fprintf(stderr, " Could not allocate pair memory on the device!!");
        }

        if(dim % 2 == 0){
            /*initializing pair matrix*/
            for (i = 0; i < n; i++)
                *(pair + i) = i;
        }
        else{
            for (i = 0; i < dim; i++)
                *(pair + i) = i;
            *(pair + n - 1) = 999;
        }
        printf("\n n %d\n", n);

        #if 1
        status = hipMemcpy(submat1, bufptr, submatsize[num]*submatsize[num]*sizeof(double), hipMemcpyHostToDevice);
        if( status != hipSuccess) {
            fprintf(stderr, " Could not copy the submat array to the host!!");
        }

        status = hipMemcpy(d_pair, pair, n*sizeof(int), hipMemcpyHostToDevice);
        if( status != hipSuccess) {
            fprintf(stderr, " Could not copy the pair array to the host!!");
        }

        jacobi<<<1, n/2>>>(submat1, d_pair, submatsize[num], d_cont, tolerance);
	//jacobi<<<1, n/2, 0, streams[cnt]>>>(submat1, d_pair, submatsize[num], d_cont, tolerance);
	//cudaDeviceSynchronize();
        //cudaStreamSynchronize(0);

        hipMemcpy(bufptr, submat1, submatsize[num]*submatsize[num]*sizeof(double), hipMemcpyDeviceToHost);

        printf("\n\n eigen values here for #%d", cnt);
    //print the 1st submatrix
        for(i=0; i< submatsize[num]; i++){
            for(j=0; j<submatsize[num]; j++){
	      if(i == j)
                  printf(" %lf", *(bufptr+i*submatsize[num]+j));
              }
            printf("\n");
        }
        num += 1;
        hipFree(d_pair);
        hipFree(d_cont);
        hipFree(submat1);
        free(bufptr);
        free(pair);
    #endif
    }

    pthread_mutex_unlock(&crit_lock);

#endif    
    hipFree(posptr);
    hipFree(colptr);
    hipFree(chargptr);

}

void *parse_pos_file(void *arg)
{
    unsigned int i = 0;
    //char *buf;
    int count = 0;
    pthread_t threads[NTHREADS];
    void * retvals[NTHREADS];
    int *cnt;

    printf(" Thread 2");
    // cond_wait
    pthread_mutex_lock(&cond_var_lock);
    while(frame_bufs[0] == 0)
        pthread_cond_wait(&cond_var, &cond_var_lock);
    pthread_mutex_unlock(&cond_var_lock);

    //printf("\n comes here --1");
    //printf("\n");
    //buf = frame_bufs[0];
    //for(i = 0; i < 100; i++)
    //    printf("%c", *(buf+i));

   
    // launch 50 threads which parses the pos frame buffer and performs cuda operations in parallel.
    // each of the thread creates the coulomb matrix, does submatrix reductions and computes eigen values.
    // the eigen values are finally written as hdf5 files.
//    int k = 0;
    for(count = 0; count < NTHREADS; count++)
    {
        fflush(stdout);
        cnt = (int *)malloc(1*sizeof(int));
        *cnt = count;
        if(pthread_create(&threads[count], NULL, main_job_cuda, (void *)cnt) != 0)
	{
	    printf("error: cannot create thread # %d\n", *cnt);
	    return (void *)NULL;
	}
    }

    for(i = 0; i < NTHREADS; i++)
    {
        if(pthread_join(threads[i], &retvals[i]) != 0)
	{
	    printf("error: cannot join thread # %d\n", i);
	    return (void *)NULL;
	}
    }

#if 0
// parser for each frame
    double frame[NUM_ATOMS][3];
    unsigned int j = 0;
    char temp1[18], temp2[18], temp3[18];
    buf = frame_bufs[0] + line1_size + line2_size;
    //i += line1_size + line2_size;
    while(j < NUM_ATOMS){
        memcpy(temp1, buf + 6, 18);
	memcpy(temp2, buf + 32, 18);
	memcpy(temp3, buf + 58, 18);
	for(i = 0; i < 18; i++)
	    printf(" %c", temp3[i]);
        printf("\n");
        sscanf(temp1, "%lf", &frame[j][0]);
	sscanf(temp2, "%lf", &frame[j][1]);
	sscanf(temp3, "%lf", &frame[j][2]);
	j += 1;
	buf += line3_size;
    }
#endif 

#if 0
    for(j = 0; j < NUM_ATOMS; j++){
        for(i = 0; i < 3; i++){
            printf(" %.17g", frame[j][i]);
	    }
	printf("\n");
    }
#endif

}

void *read_pos_file(void *pth)
{
    char *path = (char *)pth;
    FILE *fp;
    char buf[256] = {0,};
    unsigned int i = 0;
    //unsigned int frame_size = 0;
    char *buff;
    printf("\n File path is %s", path);

    fp = fopen(path, "r");

    // estimate the memory size needed for a frame from pos file. Reads the first three lines.
    while(fgets(buf, 256, (FILE *)fp) != NULL){
        //printf("strlen(buf) is %d", strlen(buf));
	i++;

	if(i == 1){
	    frame_size += strlen(buf);
	    line1_size = strlen(buf);
	}
	else if(i == 2){
	frame_size += strlen(buf);
	line2_size = strlen(buf);
	}
	else{
	    frame_size += NUM_ATOMS * strlen(buf);
	    line3_size = strlen(buf);
        }
	if(i == 3)
	    break;
    }

#if 1
    for(i = 0; i < 10; i++)
        printf("%c", buf[i]);
#endif

//    fclose(fp);
    printf("memory size req for a frame is %d", frame_size); 
    fseek(fp, 0, SEEK_SET);

    pthread_mutex_lock(&cond_var_lock);
    buff = (char *)malloc(frame_size * 50 * sizeof(char));
    fread(buff, sizeof(char), frame_size * 50, fp);
    frame_bufs[0] = buff;
    pthread_cond_signal(&cond_var);
    pthread_mutex_unlock(&cond_var_lock);

    for(i = 1; i < 10; i++)
    {
        buff = (char *)malloc(frame_size * 50 * sizeof(char));
	fread(buff, sizeof(char), frame_size * 50, fp);
        frame_bufs[i] = buff;
    }
    fclose(fp);
}


int main(int argc, char *argv[])
{
    pthread_t thread1, thread2;
    int ret1, ret2; 
    char pos_file_name[] = "pos.xyz";
    char cwd[PATH_MAX];
    char file_path[PATH_MAX + strlen(pos_file_name)];

    if(argc == 1)
    {    
        printf("Program expects the directory name where pos/frc/ener file is located");
        return -1;
    }

    if (getcwd(cwd, sizeof(cwd)) != NULL) {
        printf("Current working dir: %s\n", cwd);
    } else {
       perror("getcwd() error");
       return 1;
    }

    //printf("cwd is %s", cwd);
    strcat(file_path, cwd);
    strcat(file_path, "/");
    strcat(file_path, argv[1]);
    strcat(file_path, "/");
    strcat(file_path, pos_file_name);
    printf("\n file_path is %s", file_path);

// launch the thread which reads data from the pos file and stores in small buffers.
    ret1 = pthread_create( &thread1, NULL, read_pos_file, (void *)file_path);     
    ret2 = pthread_create( &thread2, NULL, parse_pos_file, (void *)NULL);

    pthread_join(thread1, NULL);
    pthread_join(thread2, NULL);

    return 0;
}
