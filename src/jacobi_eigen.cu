#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <stdlib.h>
#include <iostream>

__global__ void jacobi(double *arr_ptr, int *pair_arr, int n, int *cont, double tolerance)
{
    unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
    double c, s;
    unsigned int k = 0;
    unsigned int i, j;
    unsigned int p, q, temp;
    unsigned int te = (n % 2 ? n+1 : n);

    //printf("\n te %d n %d", te, n);

    while(*cont != 0)
    {
        if(id == 0)
        {
            k += 1;
            //printf("k val %d", k);
        }
        __syncthreads();

        //for(in = 0; in < n/2; in++)
        //{
            p = *(pair_arr + id);
            q = *(pair_arr + id + te/2);

            //printf("\n p & q %d %d \n", p, q);

            if(p != 999 && q != 999){
            if(p > q)
            {
                temp = q;
                q = p;
                p = temp;
            }

            if(*(arr_ptr + n*p +q) != 0)
            {
                double torque, t;
                torque = ( *(arr_ptr + q * n + q) - *(arr_ptr + p * n + p))/(2*(*(arr_ptr + p * n + q)));
                if (torque >= 0)
                    t = 1/(torque + sqrt(1+torque*torque));
                else
                    t = -1/(-torque + sqrt(1+torque*torque));

                c = 1/sqrt(1+t*t);
                s = t*c;
            }
            else
            {
                c = 1;
                s = 0;
            }

            /* A = transpose(J)*A*J */
            for (i = 0; i < n; i++)
            {
                double Api = (*(arr_ptr + p * n + i))*c + (*(arr_ptr + q * n + i))*(-s);
                double Aqi = (*(arr_ptr + p * n + i))*s + (*(arr_ptr + q * n + i))*c;
                //__syncthreads();
                *(arr_ptr + p * n + i) = Api;
                *(arr_ptr + q * n + i) = Aqi;
            }

            for (i = 0; i < n; i++)
            {
                double Aip = (*(arr_ptr + i * n + p))*c + (*(arr_ptr + i * n + q))*(-s);
                double Aiq = (*(arr_ptr + i * n + p))*s + (*(arr_ptr + i * n + q))*c;
                //__syncthreads();
                *(arr_ptr + i * n + p) = Aip;
                *(arr_ptr + i * n + q) = Aiq;
            }

            //for(i = 0; i < n; i++)
            //{
            //    for(j = 0; j < n; j++)
            //    {
                   // if(i != j)
            //{
            //         printf("  %lf", *(arr_ptr + i*n + j));
            //}
            //    }
            //    printf(" \n");
            //}

            }

        //}
        __syncthreads();

        // chess reordering
        if(id == 0)
        {
            //unsigned int te = (n % 2 ? n-1 : n);
            //unsigned int temp;
            temp = *(pair_arr + te/2 - 1);

            for(i = te/2-1; i > 1; i--)
            {
                *(pair_arr + i) = *(pair_arr + i - 1);
            }

            *(pair_arr + 1) = *(pair_arr + te/2);

            for(i = te/2; i < te-1; i++)
            {
                *(pair_arr + i) = *(pair_arr + i + 1);
            }

            *(pair_arr + te - 1) = temp;

            //printf("\n k val %d", k);
           if(k == te)
           {
               double val = 0;
               for(i = 0; i < n; i++)
               {
                   for(j = 0; j < n; j++)
                   {
                       if(i != j)
                       {
                           val += pow(*(arr_ptr + n*i + j), 2);
                       }
                   }
               }
               //printf("sqrt(val) %lf", sqrt(val));
               if(sqrt(val) <= tolerance)
                   *cont = 0;
               else
                   *cont = 1;
               k = 0;
           }
           //break;
        }
        __syncthreads();
    }
}
