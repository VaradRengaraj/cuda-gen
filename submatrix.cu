#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <stdlib.h>
#include <iostream>

#define NR 81

//#define NR 9
__device__ double* dataptr[NR];
__global__ void submatrix(double *col, int nx, int *submatsizes)
{
    unsigned int local[NR] = {0,};
    unsigned int i, j;
    unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
    unsigned int k,l;
    double *buf;
    k=0;
    l=0;

    if(id < nx)
    {

    for(i=id; i<=nx*nx; i+=nx)
    {
        if(*(col+i) != 0){
            local[l] = k+1;
            l++;
        }
        k++;
    }

   // allocate memory for the submatrix based on the size
    for(i=0; i<NR; i++)
    {
        if(local[i] == 0)
            break;
    }

    k=i;
    *(submatsizes+id) = k;
    //printf("\n k--id %d %d", k, id);
    buf = (double *)malloc(k*k*sizeof(double));
    l=0;

    for(i=0; i<k; i++)
    {
        for(j=0; j<k; j++)
        {
            *(buf+l) = *(col+(local[i]-1)*nx+(local[j]-1));
            l++;
        }
    }

    dataptr[id] = buf;
    __syncthreads();
    }
}


#if 1
__global__ void copysubmat(double *subm, int N, int num)
{
    unsigned int i;
    unsigned int id = num;//blockIdx.x*blockDim.x+threadIdx.x;
    double *bufptr = dataptr[id];

    for(i=0; i<N*N; i++)
        subm[i] = bufptr[i];
}
#endif


